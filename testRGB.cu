#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
    FILE * f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);
    
    if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);
    
    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName); 
        exit(EXIT_FAILURE); 
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height, int originalWidth, char *fileName)
{
    FILE * f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }   

    fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            int i = r * originalWidth + c;
            fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
        }
    }
    
    fclose(f);
}

__global__ void convertRgb2GrayKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	if (idx < width * height)	
	{	
		uint8_t red = inPixels[3 * idx];	
		uint8_t green = inPixels[3 * idx + 1];	
		uint8_t blue = inPixels[3 * idx + 2];	
		outPixels[idx] = 0.299f * red + 0.587f * green + 0.114f * blue;	
	}
}

void convertRgb2Gray(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width; c++)
            {
                int i = r * width + c;
                uint8_t red = inPixels[3 * i];
                uint8_t green = inPixels[3 * i + 1];
                uint8_t blue = inPixels[3 * i + 2];
                outPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
            }
        }
	}
	else // use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO: Allocate device memories
		int numPixels = width * height;	
		uint8_t *d_inPixels, *d_outPixels;	
		hipMalloc(&d_inPixels, sizeof(uint8_t) * numPixels * 3);	
		hipMalloc(&d_outPixels, sizeof(uint8_t) * numPixels);
		
		// TODO: Copy data to device memories
		hipMemcpy(d_inPixels, inPixels, sizeof(uint8_t) * numPixels * 3, hipMemcpyHostToDevice);
		
		// TODO: Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((numPixels + blockSize.x - 1) / blockSize.x);	
		
		convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);
		hipDeviceSynchronize();
		
		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, sizeof(uint8_t) * numPixels, hipMemcpyDeviceToHost));

		// TODO: Free device memories
		hipFree(d_inPixels);
		hipFree(d_outPixels);

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

float computeError(uint8_t * a1, uint8_t * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
		err += abs((int)a1[i] - (int)a2[i]);
	err /= n;
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

int main(int argc, char ** argv)
{	
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input RGB image file
	int numChannels, width, height;
	uchar3 * inPixels;
	int desiredWidth;

	readPnm(argv[1], numChannels, width, height, inPixels);
	if (numChannels != 3)
		return EXIT_FAILURE; // Input image must be RGB
	printf("Image size (width x height): %i x %i\n\n", width, height);
	desiredWidth = atoi(argv[3]);

	// Convert RGB to grayscale not using device
	uchar3 * correctOutPixels= (uchar3 *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);

	// Convert RGB to grayscale using device
	uchar3 * outPixels= (uchar3 *)malloc(width * height);
	dim3 blockSize(32, 32); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	} 
	convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize); 

	// Compute mean absolute error between host result and device result
	float err = computeError(outPixels, correctOutPixels, width * height);
	printf("Error between device result and host result: %f\n", err);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, 1, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixels, 1, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(correctOutPixels);
	free(inPixels);
	free(outPixels);
}
